#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include "PointCloud.h"
#include <limits>
#include <algorithm>
#include "hipblas.h"
#include "hip/hip_math_constants.h"

/**
 * Given the two points: p1, p2 and axis info represented by axisOrigin and direction
 * x, y, z, calculate the score of the grasp
 * All points and axis info should be in the same frame
 */
 __device__
 float dotProduct(float3 v1, float3 v2){
     return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z; 
 }
 
 __device__
 float3 crossProduct(float3 v1, float3 v2){
     float x = v1.y * v2.z - v1.z * v2.y;
     float y = v1.z * v2.x - v1.x * v2.z;
     float z = v1.x * v2.y - v1.y * v2.x;
 
     return make_float3(x, y, z);
 }

 __device__
 float norm(float3 v){
     return norm3df(v.x, v.y, v.z);
 }

 __device__
 float3 operator+(const float3 &a, const float3 &b){
     return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
 }

 __device__
 float3 operator/(const float3 &a, const float &b){
     return make_float3(a.x / b, a.y / b, a.z / b);
 }

 __device__
 float3 operator*(const float3 &a, const float &b){
     return make_float3(a.x * b, a.y * b, a.z * b);
 }


/**
 * points: points with generated grasps
 * pointnum: size of poitns
 * threshold: each point should have at least this number of neighbors with valid grasps
 */
__global__
void filterGraspsByNeighborBrute(Point* points, int point_num, float threshold){
    // In a brute force way, it is an O(k^2*n^2) algorithm, looping through all grasps, then testing all neighbors' grasps
    int point_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride      = blockDim.x * gridDim.x;

    for (int i = point_index; i < point_num; i += stride){
        Point& curr = points[i];

        int normalSimilarGrasps = 0;
        int neighborNum = 0;
        for (int j = 0; j < point_num; j++) {
            Point& another = points[j];

            float diff_x, diff_y, diff_z;
            diff_x = another.x - curr.x;
            diff_y = another.y - curr.y;
            diff_z = another.z - curr.z;

            if (norm3df(diff_x, diff_y, diff_z) > 0.01) continue;

            neighborNum += 1;
            for (int currIter = 0; currIter < curr.generated_grasp; currIter++){
                for (int anotherIter = 0; anotherIter < another.generated_grasp; anotherIter++){
                    Point& currGraspPair    = points[curr.antiPoints[currIter]];
                    Point& anotherGraspPair = points[another.antiPoints[anotherIter]];
                    float3 currGraspDir     = make_float3(currGraspPair.x-curr.x, currGraspPair.y-curr.y, currGraspPair.z-curr.z);
                    float3 anotherGraspDir  = make_float3(anotherGraspPair.x-another.x, anotherGraspPair.y-another.y, anotherGraspPair.z-another.z);

                    // Count the grasp as a back-up if the cosine between them is larger than 0.7
                    if (dotProduct(currGraspDir, anotherGraspDir) / (norm(currGraspDir) * norm(anotherGraspDir)) > 0.9){
                        normalSimilarGrasps += 1;
                    }
                }
            }
        }        

        // curr.filteredGraspNum = normalSimilarGrasps > threshold ? curr.generated_grasp : 0;
        if (normalSimilarGrasps < threshold) {
            curr.filteredGraspNum = 0;
        } 
    }
}

__device__
// Return the direction if there is a direction without collision, or return (0, 0, 0)
float3 collidedWithGripper(Point* points, int size, int ind1, int ind2, float aabbInnerRadius, float aabbOuterRadius, float gripperHeight){
    Point& p1 = points[ind1];
    Point& p2 = points[ind2];

    // Find the mid point between two contacts
    float3 mid  = make_float3((p1.x + p2.x) / 2, (p1.y + p2.y) / 2, (p1.z + p2.z) / 2);
    float3 zAxis= make_float3(p2.x - p1.x, p2.y - p1.y, p2.z - p1.z);
    // Define a grasp coordinate, with mid-point as origin, grasp axis (p1-p2) as positive z, arbitrarily
    // define an axis as x, and calculate y
    float3 temp = make_float3(zAxis.x + 1, zAxis.y, zAxis.z);
    if (zAxis.y < 0.0000001 && zAxis.z == 0.0000001){
        temp.y = temp.y + 1;
    }
    float3 xAxis= crossProduct(zAxis, temp);

    float3 yAxis= crossProduct(zAxis, xAxis);
    
    xAxis = xAxis / norm(xAxis);
    yAxis = yAxis / norm(yAxis);
    zAxis = zAxis / norm(zAxis);

    // Loop through all points and find if at least one slice is free of collision
    bool collisionSlice[10];
    // bool collisionSliceInner[4];
    for (int i = 0; i < 10; i++){
        collisionSlice[i] = false;
    }
    for (int i = 0; i < size; i++){
        // Calculate the length from point to the axis
        Point& p = points[i];
        float3 pToMid = make_float3(p.x - mid.x, p.y - mid.y, p.z - mid.z);

        float pToMidOnX = dotProduct(pToMid, xAxis);
        float pToMidOnY = dotProduct(pToMid, yAxis);

        // If the point is out of outer radius, it won't collide with the gripper
        if (norm3df(pToMidOnX, pToMidOnY, 0) > aabbOuterRadius){
            continue;
        }

        float pToMidOnZ = dotProduct(pToMid, zAxis);
        pToMidOnZ = abs(pToMidOnZ);
        if (pToMidOnZ > gripperHeight) {
            // If the point is too far from the grasp, ignore it
            continue;
        }

        // If the point is inside of the inner radius, check whether it collides with the finger, by calculating the
        // distance from origin along axis direction. It has to be within the distance of two contact points
        if (norm3df(pToMidOnX, pToMidOnY, 0) < aabbInnerRadius){
            // float checkRadius = 0.01;
            float epsilon     = 0.001;
            float dist_between_contacts = norm(make_float3(p2.x-p1.x, p2.y-p1.y, p2.z-p1.z));
            if ( pToMidOnZ < (dist_between_contacts/2 + epsilon) ) continue;

            float dist1 = norm3df(p.x - p1.x, p.y - p1.y, p.z - p1.z);
            float dist2 = norm3df(p.x - p2.x, p.y - p2.y, p.z - p2.z);

            if (dist1 > gripperHeight && dist2 > gripperHeight) continue;
        }

        // Divide into 10 regions, if one of them is collision-free, treat the grasp as valid
        // This number '10' should vary according to the depth of the gripper's collision body and the distance from
        // the collision body to the grasp center
        float cosAngleX = pToMidOnX / norm3df(pToMidOnX, pToMidOnY, 0);

        float angle = acosf(cosAngleX);
        if (pToMidOnY < 0) angle = angle + HIP_PI_F;
        angle = angle >= 2 * HIP_PI_F ? 0 : angle;

        collisionSlice[(int) (10 * angle / (2 * HIP_PI_F))] = true;
    }

    int noCollisionSlice = -1;
    for (int i = 0; i < 10; i++){
        if ( collisionSlice[i] == false ){
            noCollisionSlice = i;
            break;
        }
    }
    if (noCollisionSlice == -1) return make_float3(0, 0, 0);
    else {
        float angle = ((2 * HIP_PI_F) * noCollisionSlice) / 10;
        float3 dir  = ( xAxis * cosf(angle) + yAxis * sinf(angle) );
        return dir;
    }
}

// util functions for generating and filtering grasps
/**
 * aabbInnerRadius and aabbOuterRadius defines a revoluted body generated by an AABB rotating along the
 * grasp axis (axis connecting two end effectors).
 */
__global__
void generateGraspBrute(Point* points, float friction_coef, float jaw_span, int point_num, int candidateNum,
    float aabbInnerRadius, float aabbOuterRadius, float gripperHeight){
    int point_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride      = blockDim.x * gridDim.x;
    for (int i = point_index; i < point_num; i+=stride){
        Point& curr = points[i];
        // Brute force loop through all points and check
        for (int j = i+1; j < point_num; j++){
            Point& another =  points[j];

            float diff_x, diff_y, diff_z;
            diff_x = another.x - curr.x;
            diff_y = another.y - curr.y;
            diff_z = another.z - curr.z;

            float square_norm = diff_x * diff_x + diff_y * diff_y + diff_z * diff_z;
            if (diff_x * diff_x + diff_y * diff_y + diff_z * diff_z > jaw_span * jaw_span){
                // too far for the jaw
                continue;
            }

            float angle1     = -diff_x * curr.nx - diff_y * curr.ny - diff_z * curr.nz;
            float cos_angle1 = angle1 / sqrtf(square_norm);
            float tan_angle1 = tanf(acosf(cos_angle1));

            float angle2     = diff_x * another.nx + diff_y * another.ny + diff_z * another.nz;
            float cos_angle2 = angle2 / sqrtf(square_norm);
            float tan_angle2 = tanf(acosf(cos_angle2));

            if ( cos_angle1 < 0.0001 || tan_angle1 > friction_coef || cos_angle2 < 0.0001 || tan_angle2 > friction_coef) {
                // Out of friction cone
                continue;
            } 
            else {
                float3 noCollisionDir = collidedWithGripper(points, point_num, i, j, aabbInnerRadius, aabbOuterRadius, gripperHeight);
                if (norm(noCollisionDir) < 0.01){
                    continue;
                }
                else {
                    // Use the maximum angle as score -- the larger the worse
                    float score = tan_angle1 > tan_angle2 ? tan_angle1 : tan_angle2;
                    curr.addAntipodal(j, score, candidateNum, noCollisionDir);
                }
            }
        }
    }
}

__host__ __device__
void Point::addAntipodal(int index, float score, int candidateNum, float3 noCollisionDir){
    // Manage the priority queue here
    // Score is the larger angle
    // printf("min score: %f, current score: %f\n", this->worst_score, score);

    if (generated_grasp == 0){
        this->antiPoints[this->generated_grasp] = index;
        this->scores    [this->generated_grasp] = score;
        this->directions[this->generated_grasp] = noCollisionDir;
        this->generated_grasp += 1;

        this->worst_score = score;
    }

    else if (this->generated_grasp < candidateNum) {
        this->antiPoints[this->generated_grasp] = index;
        this->scores    [this->generated_grasp] = score;
        this->directions[this->generated_grasp] = noCollisionDir;
        this->generated_grasp += 1;

        this->worst_score = score > this->worst_score ? score : this->worst_score;
    }

    else if (score < this->worst_score){
        float new_worst_score = score;
        for (int i = 0; i < candidateNum; i++){
            if (this->scores[i] == this->worst_score) {
                // Insert point to replace worst score
                this->scores[i] = score;
                this->antiPoints[i] = index;
                this->directions[i] = noCollisionDir;
            } else {
                // Find new worst score
                new_worst_score = this->scores[i] > new_worst_score ? this->scores[i] : new_worst_score;
            }
        }

        this->worst_score = new_worst_score;
    }

    this->filteredGraspNum = this->generated_grasp;
}

Point::~Point(){
    if (this->antiPoints != nullptr)
        hipFree(this->antiPoints);
    if (this->scores != nullptr)
        hipFree(this->scores);
    if (this->directions != nullptr)
        hipFree(this->directions);
}

PointCloud::PointCloud(Point* points, int size, int candidateNum){
    this->size = size;
    this->candidateNum = candidateNum;
    // 1. constructor: put points on to cuda memory
    hipMallocManaged(&(this->cloud), size * sizeof(Point));
    hipMemcpy(this->cloud, points, size * sizeof(Point), hipMemcpyHostToDevice);
    for (int i = 0; i < size; i++){
        // Allocate space for 10 points to store antipodal grasp
        hipMallocManaged( &(this->cloud[i].antiPoints), candidateNum * sizeof(int));
        hipMallocManaged( &(this->cloud[i].scores), candidateNum * sizeof(float));
        hipMallocManaged( &(this->cloud[i].directions), candidateNum * sizeof(float3));
    }

    // 2. Create grid depending on points distribution
    this->xMin = std::numeric_limits<float>::max();
    this->yMin = std::numeric_limits<float>::max();
    this->zMin = std::numeric_limits<float>::max();
    this->xMax = std::numeric_limits<float>::min();
    this->yMax = std::numeric_limits<float>::min();
    this->zMax = std::numeric_limits<float>::min();
    
    for (int i = 0; i < size; i++){
        Point& p = points[i];

        this->xMin = std::min(this->xMin, p.x);
        this->yMin = std::min(this->yMin, p.y);
        this->zMin = std::min(this->zMin, p.z);

        this->xMax = std::max(this->xMax, p.x);
        this->yMax = std::max(this->yMax, p.y);
        this->zMax = std::max(this->zMax, p.z);
    }

    // // One grid per 0.001 (1 mm)
    // this->xDim = (int) ((this->xMax - this->xMin) / GRID_STRIDE + 1);
    // this->yDim = (int) ((this->yMax - this->yMin) / GRID_STRIDE + 1);
    // this->zDim = (int) ((this->zMax - this->zMin) / GRID_STRIDE + 1);

    // hipMallocManaged(&(this->grid), this->xDim * this->yDim * this->zDim * sizeof(Cell));
    
    std::cout << "Populated points: " << size << std::endl;
}

void PointCloud::generateGraspsBrute(float friction_coef, float jaw_span){
    generateGraspBrute<<<128, 256>>>(this->cloud, friction_coef, jaw_span, this->size, this->candidateNum, 0.1, 0.2, 0.2);
    hipDeviceSynchronize();
}

// Filter grasps, such that neighbors of current points should also have valid grasp
void PointCloud::filterGraspsByNeighbor(){
    filterGraspsByNeighborBrute<<<128, 256>>>(this->cloud, this->size, 1000);
    hipDeviceSynchronize();
}

PointCloud::~PointCloud(){
    hipFree(this->cloud);
}
